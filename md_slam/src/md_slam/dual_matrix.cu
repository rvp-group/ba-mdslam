#include "hip/hip_runtime.h"
// Copyright 2022 Luca Di Giammarino
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice,
//    this list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its contributors
//    may be used to endorse or promote products derived from this software
//    without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.

#include "dual_matrix.cuh"

namespace md_slam {
  template <typename CellType_>
  __global__ void
  fill_kernel(CellType_* data_device_, const CellType_ value_, const size_t capacity_) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < capacity_)
      data_device_[tid] = value_;
  }

  template <typename CellType_>
  void DualMatrix_<CellType_>::fill(const CellType_& value_, const bool device_only_) {
    fill_kernel<<<_n_blocks, _n_threads>>>(_buffers[Device], value_, _capacity);
    CUDA_CHECK(hipDeviceSynchronize());
    if (device_only_)
      return;
    // fill host and then fill device
    for (int i = 0; i < _capacity; ++i) {
      _buffers[Host][i] = value_;
    }
  }

} // namespace md_slam
